#include <iostream>
#include <hip/hip_runtime.h>


// CUDA core for AES algorithm
__global__ void aesEncryptionKernel(unsigned char *data, unsigned char *key, int dataSize) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < dataSize) {
        data[i] = data[i] ^ key[i % 16];
    }
}

void generateData(unsigned char *data, int dataSize) {
    data[0] = rand() % 256;
    for (int i = 1; i < dataSize; ++i) {
        data[i] = (unsigned char) (data[i - 1] + i) % 256;
    }
}

int main(int argc, char *argv[]) {
    int dataSize = atoi(argv[1]); // unhandled exception if no argument is passed
    unsigned char *data = (unsigned char *) malloc(dataSize);
    unsigned char *key = (unsigned char *) malloc(16);

    generateData(data, dataSize);
    generateData(key, 16);

    unsigned char *d_data, *d_key;
    hipMalloc(&d_data, dataSize);
    hipMalloc(&d_key, 16);

    // copy data to device
    hipMemcpy(d_data, data, dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, 16, hipMemcpyHostToDevice);

    // variables to track time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    int repeat = 10;

    for (int i = 0; i < repeat; i++)
        aesEncryptionKernel<<<(dataSize + 255) / 256, 256>>>(d_data, d_key, dataSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // copy data back to host
    hipMemcpy(data, d_data, dataSize, hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Time taken: " << milliseconds / repeat << " ms" << std::endl;
}